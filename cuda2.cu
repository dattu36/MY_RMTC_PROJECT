#include "hip/hip_runtime.h"
%%writefile cuda_example.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call) \
do { \
    hipError_t cudaErr = (call); \
    if (cudaErr != hipSuccess) { \
        fprintf(stderr, "CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cudaErr)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call) \
do { \
    hipError_t cudaErr = (call); \
    if (cudaErr != hipSuccess) { \
        fprintf(stderr, "CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cudaErr)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

__global__ void pigeonhole_sort_kernel(int* arr, int n, int min_val, int range_size) {
    extern __shared__ int local_pigeonholes[];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Initialize local pigeonholes
    for (int i = threadIdx.x; i < range_size; i += blockDim.x) {
        local_pigeonholes[i] = 0;
    }
    __syncthreads();

    // Count occurrences in local pigeonholes
    while (tid < n) {
        atomicAdd(&local_pigeonholes[arr[tid] - min_val], 1);
        tid += stride;
    }
    __syncthreads();

    // Perform parallel prefix sum (scan) to calculate starting indices
    for (int i = 1; i < range_size; i <<= 1) {
        if (threadIdx.x >= i) {
            local_pigeonholes[threadIdx.x] += local_pigeonholes[threadIdx.x - i];
        }
        __syncthreads();
    }

    // Write sorted values to shared memory in correct order
    tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n) {
        int index = atomicSub(&local_pigeonholes[arr[tid] - min_val], 1) - 1;
        arr[index] = arr[tid];
        tid += stride;
    }
}


int main() {
    hipEvent_t sort_start, sort_end;
    float sort_time;

    int n;
    int* arr = NULL;

    CUDA_CHECK(hipEventCreate(&sort_start));
    CUDA_CHECK(hipEventCreate(&sort_end));
    int capacity = 100000;
    arr = (int*)malloc(capacity * sizeof(int));

    FILE* file = fopen("/content/drive/MyDrive/T10I4D100K.dat.txt", "r");
    if (file == NULL) {
        fprintf(stderr, "Could not open the file for reading.\n");
        free(arr);
        return 1;
    }

    printf("Numbers read from the file: ");
    int count = 0;

    while (fscanf(file, "%d", &arr[count]) != EOF) {
        count++;

        if (count >= capacity) {
            // printf("Too many integers in the file. Increase the array size if necessary.\n");
            break;
        }
    }

    fclose(file);

    n = count; // Set the total number of elements.

    // Copy data to GPU
    int* d_arr;
    CUDA_CHECK(hipMalloc((void**)&d_arr, n * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice));

    int range_size = 100000;  // Adjust this value based on your data range
    int min_val = 0;          // Adjust this value based on your data range

    // Calculate block and grid dimensions
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    CUDA_CHECK(hipEventRecord(sort_start));
    pigeonhole_sort_kernel<<<grid_size, block_size, range_size * sizeof(int)>>>(d_arr, n, min_val, range_size);
      CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(sort_end));
    CUDA_CHECK(hipEventSynchronize(sort_end));
    CUDA_CHECK(hipEventElapsedTime(&sort_time, sort_start, sort_end));


    printf("\nSorting function execution time: %f seconds\n", sort_time / 1000.0);

    // Copy sorted data back to the host
    CUDA_CHECK(hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost));
    printf("\nSorted Data: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");
    CUDA_CHECK(hipDeviceSynchronize());

    // You can now use the arr for further processing or output.
    // Dont forget to free any dynamically allocated memory.
    free(arr);
    CUDA_CHECK(hipFree(d_arr));

    return 0;
}

